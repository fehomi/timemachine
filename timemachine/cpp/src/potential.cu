#include <iostream>

#include "potential.hpp"
#include "gpu_utils.cuh"
#include "surreal.cuh"

namespace timemachine {

void Potential::execute_host(
    const int N,
    const int P,
    const double *h_x, // [N,3]
    const double *h_p, // [P,]
    const double *h_box, // [3, 3]
    const double lambda, // [1]
    unsigned long long *h_du_dx, // [N,3]
    double *h_du_dp, // [P]
    double *h_du_dl, //
    double *h_u) {

    double *d_x;
    double *d_p;
    double *d_box;

    const int D = 3;

    gpuErrchk(hipMalloc(&d_x, N*D*sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N*D*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_p, P*sizeof(double)));
    gpuErrchk(hipMemcpy(d_p, h_p, P*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D*D*sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D*D*sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx; // du/dx
    double *d_du_dp;
    double *d_du_dl; // du/dl
    double *d_u = nullptr; // u

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMalloc(&d_du_dx, N*D*sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_du_dx, 0, N*D*sizeof(unsigned long long)));
    gpuErrchk(hipMalloc(&d_du_dp, P*sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_du_dp, 0, P*sizeof(unsigned long long)));
    gpuErrchk(hipMalloc(&d_du_dl, sizeof(double)));
    gpuErrchk(hipMemset(d_du_dl, 0, sizeof(double)));
    gpuErrchk(hipMalloc(&d_u, sizeof(double)));
    gpuErrchk(hipMemset(d_u, 0, sizeof(double)));


    this->execute_device(
        N,
        P,
        d_x, 
        d_p,
        d_box,
        lambda,
        d_du_dx,
        d_du_dp,
        d_du_dl,
        d_u,
        static_cast<hipStream_t>(0)
    );

    gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N*D*sizeof(*h_du_dx), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dx));
    gpuErrchk(hipMemcpy(h_du_dp, d_du_dp, P*sizeof(*h_du_dp), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dp));
    gpuErrchk(hipMemcpy(h_du_dl, d_du_dl, sizeof(*h_du_dl), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dl));
    gpuErrchk(hipMemcpy(h_u, d_u, sizeof(*h_u), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_u));
    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_p));
    gpuErrchk(hipFree(d_box));

};



void Potential::execute_host_du_dx(
    const int N,
    const int P,
    const double *h_x, // [N,3]
    const double *h_p, // [P,]
    const double *h_box, // [3, 3]
    const double lambda, // [1]
    unsigned long long *h_du_dx) {

    double *d_x;
    double *d_p;
    double *d_box;

    const int D = 3;

    gpuErrchk(hipMalloc(&d_x, N*D*sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N*D*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_p, P*sizeof(double)));
    gpuErrchk(hipMemcpy(d_p, h_p, P*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D*D*sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D*D*sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx; // du/dx

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMalloc(&d_du_dx, N*D*sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_du_dx, 0, N*D*sizeof(unsigned long long)));

    this->execute_device(
        N,
        P,
        d_x, 
        d_p,
        d_box,
        lambda,
        d_du_dx,
        nullptr,
        nullptr,
        nullptr,
        static_cast<hipStream_t>(0)
    );

    gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N*D*sizeof(*h_du_dx), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dx));
    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_p));
    gpuErrchk(hipFree(d_box));

};

}

