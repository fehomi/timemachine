#include "context.hpp"
#include "gpu_utils.cuh"
#include <iostream>
#include <chrono>

namespace timemachine {

Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    Integrator* intg,
    std::vector<BoundPotential *> bps) :
    N_(N),
    intg_(intg),
    bps_(bps),
    step_(0) {

    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N*3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N*3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3*3);

    gpuErrchk(hipMalloc(&d_du_dx_t_, N*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMalloc(&d_u_t_, 1*sizeof(*d_u_t_)));

    // for(int i=0; i < bps.size(); i++) {
    //     hipStream_t stream;
    //     gpuErrchk(hipStreamCreate(&stream));
    //     streams_.push_back(stream);
    // }


};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_u_t_));
    gpuErrchk(hipFree(d_du_dx_t_));

    // for(int i=0; i < streams_.size(); i++) {
    //     gpuErrchk(hipStreamDestroy(streams_[i]));
    // }
};

void Context::add_observable(Observable *obs) {
    this->observables_.push_back(obs);
}

void Context::step(double lambda) {

    // the observables decide on whether or not to act on given
    // data (cheap pointers in any case)
    for(int i=0; i < observables_.size(); i++) {
        observables_[i]->observe(
            step_,
            N_,
            d_x_t_,
            d_box_t_,
            lambda
        );
    }

    gpuErrchk(hipMemset(d_du_dx_t_, 0, N_*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMemset(d_u_t_, 0, 1*sizeof(*d_du_dx_t_)));

    auto start = std::chrono::high_resolution_clock::now();

    for(int i=0; i < bps_.size(); i++) {
        bps_[i]->execute_device(
            N_,
            d_x_t_,
            d_box_t_,
            lambda,
            d_du_dx_t_,
            nullptr,
            nullptr,
            d_u_t_, // energies aren't really needed tbh
            static_cast<hipStream_t>(0) // TBD: parallelize me!
            // streams_[i]
        );
    }

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    // std::cout << "Forces time: " << elapsed.count() << "ms\n";

    // for(int i=0; i < streams_.size(); i++) {
        // gpuErrchk(hipStreamSynchronize(streams_[i]));
    // }

    intg_->step_fwd(
        d_x_t_,
        d_v_t_,
        d_du_dx_t_,
        d_box_t_
    );

    hipDeviceSynchronize();

    step_ += 1;

};


int Context::num_atoms() const {
    return N_;
}

double Context::get_u_t() const {
    double u;
    gpuErrchk(hipMemcpy(&u, d_u_t_, 1*sizeof(*d_u_t_), hipMemcpyDeviceToHost));
    return u;
}

void Context::get_du_dx_t(unsigned long long *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_du_dx_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

}